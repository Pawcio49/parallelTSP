#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>



typedef struct FillCData {
    int subsetSize;
    int ***C;
    int **dist;
    int CSize;
    int n;
    int dimension0C;
} fillCData;


typedef struct REC {
    int v;
    struct REC * prev;
} rec;


int myPow(int base, int power){
    int result = 1;
    int i;
    #pragma omp parallel for reduction(*:result) private(i)
    for(i = 0; i<power; i++){
        result *= base;
    }
    return result;
}


void print_subset(int *subset, int size) {
    printf("{");
    for (int i = 0; i < size; i++) {
        printf("%d", subset[i]);
        if (i < size - 1) {
            printf(", ");
        }
    }
    printf("}\n");
}


// Funkcja uruchamiana na GPU
__global__ void calculateCosts(int* subset, int subsetSize, int* res, int prev, int k, int* C, int* dist, int n, int dimension0C) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;

    if (m < subsetSize && subset[m] != subset[k]) {
        int cost = C[subset[m]] + dist[subset[m] * n + subset[k]];
        // int cost = C[prev * dimension0C + subset[m] * n];
        // int cost = C[8*15 + 3*4];
        // int cost = n;
        // int cost = dist[subset[m] * n + subset[k]];
        // int cost = subset[m] * n + subset[k];
        atomicMin(&res[0], cost);
        if (cost == res[0]) {
            res[1] = subset[m];
        }
    }
}


void fillC (rec * x, fillCData data) {
    int *subset = (int *)malloc(data.subsetSize * sizeof(int));
    int i = 0;
    while (x) { 
        if(x->v > 0){
            subset[i] = x->v;
            i++;
        }
        x = x -> prev;
    }
    
    int bits = 0;
    for(int i=0; i<data.subsetSize; i++){ //CUDA lub OpenMP
        bits |= 1 << subset[i];
    }

    int* flatDist = new int[data.n*data.n];
    for (int i = 0; i < data.n; i++) {
        for (int j = 0; j < data.n; j++) {
            flatDist[i * data.n + j] = data.dist[i][j];
        }
    }

    
    
    

    for(int k=0; k<data.subsetSize; k++) { // tu bez zrownoleglenia, bo musi byc po kolei
        int prev = bits & ~(1 << subset[k]);

        int res[2] = {99999,99999};
        
        int* d_subset;
        int* d_res;
        int* d_C;
        int* d_dist;

        // Alokacja pamięci na GPU
        hipMalloc((void**)&d_subset, data.subsetSize * sizeof(int));
        hipMalloc((void**)&d_res, 2 * sizeof(int));
        hipMalloc((void**)&d_C, data.n * sizeof(int));
        hipMalloc((void**)&d_dist, data.n * data.n * sizeof(int));
        // printf("C2 = %d\n", data.C[132]);
        // Przesyłanie danych z hosta do GPU
        hipMemcpy(d_subset, subset, data.subsetSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_res, res, 2 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_C, data.C[prev][0], data.n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, flatDist, data.n * data.n * sizeof(int), hipMemcpyHostToDevice);
        
         // Konfiguracja rozmiaru bloków i siatki
        int threadsPerBlock = data.subsetSize;
        int blocksPerGrid = (data.subsetSize + threadsPerBlock - 1) / threadsPerBlock;

        // Wywołanie funkcji na GPU
        calculateCosts<<<blocksPerGrid, threadsPerBlock>>>(d_subset, data.subsetSize, d_res, prev, k, d_C, d_dist, data.n, data.dimension0C);

        // Przesyłanie wyników z GPU do hosta
        hipMemcpy(res, d_res, 2 * sizeof(int), hipMemcpyDeviceToHost);
        // printf("%d\n", res[0]);
        // Zwolnienie pamięci na GPU
        hipFree(d_subset);
        hipFree(d_res);
        hipFree(d_C);
        hipFree(d_dist);

        data.C[bits][0][subset[k]] = res[0];
        data.C[bits][1][subset[k]] = res[1];
    }
    // printf("%d\n", omp_get_thread_num());
    free(subset);
}
     
void generateCombinationsAndFillC(rec * x, int level, int k, fillCData data) {
    rec X1,X2;
    if (level==0) {
        fillC(x, data);
    } 
    else 
    {
        #pragma omp task
        { 
            if (level>k) {
                X1.prev = x;
                X1.v = 0;
                generateCombinationsAndFillC(&X1,level-1, k, data);
            } 
        }
        #pragma omp task 
        {
            if (k>0) {
                X2.prev = x;
                X2.v = level;
                generateCombinationsAndFillC(&X2,level-1, k-1, data);
            }
        }
        #pragma omp taskwait  
    }
}


int TSP(int n, int **dist, int *path)
{
    int dimension0C = myPow(2, n) - 1;
    int CSize = dimension0C * n * 2;
    int ***C;
    C = (int ***)malloc(CSize * sizeof(int));
    for(int i=0; i<dimension0C;i++) {
        C[i] = (int **)malloc(n * 2 * sizeof(int));
        for(int j=0; j<2; j++) {
            C[i][j] = (int *)malloc(n * sizeof(int));
        }
    }

    for(int i=1; i<n; i++) { //CUDA - sprobowac  - raczej nie ma sensu
        C[1<<i][0][i] = dist[0][i];
        C[1<<i][1][i] = 0;
    }

    // printf("C = %d\n", C[8*15 + 3*4]);
    // printf("CSize = %d\n", CSize);

    fillCData data;
    data.C = C;
    data.dist = dist;
    data.CSize = dimension0C * n * 2;
    data.n = n;
    data.dimension0C = dimension0C;
    for(int subsetSize=2; subsetSize<n; subsetSize++){
        data.subsetSize = subsetSize;
        #pragma omp parallel
        #pragma omp single
        generateCombinationsAndFillC(NULL, n-1, subsetSize, data);
    }

    // We're interested in all bits but the least significant (the start state)
    int bits = (myPow(2, n) - 1) - 1;

    // // Calculate optimal cost
    int opt = -1;
    int parent;
    for(int k=1; k<n; k++)  { //CUDA lub OpenMP - raczej nie ma sensu
        int cost = C[bits][0][k] + dist[k][0];
        if(opt== -1 || cost<opt){
            opt = cost;
            parent = k;
        }
    }

    for(int i=n-1; i>0; i--) { // tu bez zrownoleglenia, bo musi byc po kolei
        path[i] = parent;
        int new_bits = bits & ~(1 << parent);
        parent = C[bits][1][parent];
        bits = new_bits;
    }

    path[0] = 0;
    free(C);
    return opt;
}

int readMatrix(int size, int **a, const char* filename)
{
    FILE *pf;
    pf = fopen (filename, "r");
    if (pf == NULL)
        return 0;

    for(int i = 0; i < size; ++i)
    {
        for(int j = 0; j < size; ++j) {
            
            fscanf(pf, "%d", a[i] + j);
        }
    }


    fclose (pf);
    return 1;
}

int generateMatrix(int size, int **a)
{
    srand(time(NULL)); 
    int min = 1;
    int max = 1000;
    for(int i = 0; i < size; ++i)
    {
        for(int j = 0; j < size; ++j) {
            if(i==j)
                a[i][j] = 0;
            else
                a[i][j] = rand() % (max - min + 1) + min;
        }
    }
    return 1;
}


int main(int argc, char *argv[])
{
    if(argc<3){
        printf("Type ./filename matrix_size file_with_matrix");
        return -1;
    }

    const int N = atoi(argv[1]);
    int final_path[N];

    // Adjacency matrix for the given graph
    int **adj;
    adj = (int **)malloc(N * N * sizeof(int));
    for(int i=0;i<N;i++){
        adj[i] = (int *)malloc(N * sizeof(int));
    }
    readMatrix(N, adj, argv[2]);
    // generateMatrix(N,adj);
    clock_t start, end;
    double cpu_time_used;

    start = clock();
    int final_res = TSP(N, adj, final_path);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time : %f\n", cpu_time_used);
    printf("Minimum cost : %d\n", final_res);
    printf("Path Taken : ");
    for (int i=0; i<N; i++)
        printf("%d ", final_path[i]);
    printf("\n");
    free(adj);
    return 0;
}