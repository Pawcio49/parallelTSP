#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>



typedef struct FillCData {
    int subsetSize;
    int ***C;
    int CSize;
    int n;
} fillCData;


typedef struct REC {
    int v;
    struct REC * prev;
} rec;


int myPow(int base, int power){
    int result = 1;
    int i;
    #pragma omp parallel for reduction(*:result) private(i)
    for(i = 0; i<power; i++){
        result *= base;
    }
    return result;
}


void print_subset(int *subset, int size) {
    printf("{");
    for (int i = 0; i < size; i++) {
        printf("%d", subset[i]);
        if (i < size - 1) {
            printf(", ");
        }
    }
    printf("}\n");
}


__device__ int global_dist[1000]; 


__global__ void process_data(int* input_data, int data_size) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thread_id < data_size) {
        global_dist[thread_id] = input_data[thread_id];
    }
}


__global__ void calculateCosts(int* subset, int subsetSize, int* res, int k, int* CPrev0, int n) {
    int m = threadIdx.x;

    if (m < subsetSize && m != k) {
        int cost = CPrev0[subset[m]] + global_dist[subset[m] * n + subset[k]];
        atomicMin(&res[0], cost);
        if (cost == res[0]) {
            res[1] = subset[m];
        }
    }
}


void fillC (rec * x, fillCData data) {
    int *subset = (int *)malloc(data.subsetSize * sizeof(int));
    int i = 0;
    while (x) { 
        if(x->v > 0){
            subset[i] = x->v;
            i++;
        }
        x = x -> prev;
    }
    
    int bits = 0;
    for(int i=0; i<data.subsetSize; i++){
        bits |= 1 << subset[i];
    }

    for(int k=0; k<data.subsetSize; k++) {
        int prev = bits & ~(1 << subset[k]);

        int res[2] = {999999,0};
        
        int* d_subset;
        int* d_res;
        int* d_C;

        
        hipMalloc((void**)&d_subset, data.subsetSize * sizeof(int));
        hipMalloc((void**)&d_res, 2 * sizeof(int));
        hipMalloc((void**)&d_C, data.n * sizeof(int));
        
        hipMemcpy(d_subset, subset, data.subsetSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_res, res, 2 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_C, data.C[prev][0], data.n * sizeof(int), hipMemcpyHostToDevice);
        
        int threadsPerBlock = data.subsetSize;
        int gridSize = 1;

        calculateCosts<<<gridSize, threadsPerBlock>>>(d_subset, data.subsetSize, d_res, k, d_C, data.n);

        hipMemcpy(res, d_res, 2 * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(d_subset);
        hipFree(d_res);
        hipFree(d_C);

        data.C[bits][0][subset[k]] = res[0];
        data.C[bits][1][subset[k]] = res[1];
    }
    // printf("%d\n", omp_get_thread_num());
    free(subset);
}
     
void generateCombinationsAndFillC(rec * x, int level, int k, fillCData data) {
    rec X1,X2;
    if (level==0) {
        fillC(x, data);
    } 
    else 
    {
        #pragma omp task
        { 
            if (level>k) {
                X1.prev = x;
                X1.v = 0;
                generateCombinationsAndFillC(&X1,level-1, k, data);
            } 
        }
        #pragma omp task 
        {
            if (k>0) {
                X2.prev = x;
                X2.v = level;
                generateCombinationsAndFillC(&X2,level-1, k-1, data);
            }
        }
        #pragma omp taskwait  
    }
}


int TSP(int n, int *dist, int *path)
{
    int data_size = n*n;   

    int* d_input_data;
    hipMalloc((void**)&d_input_data, data_size * sizeof(int));
    
    hipMemcpy(d_input_data, dist, data_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = data_size;
    int grid_size = 1;
    process_data<<<grid_size, threadsPerBlock>>>(d_input_data, data_size);

    int dimension0C = myPow(2, n) - 1;
    int CSize = dimension0C * n * 2;
    int ***C;
    C = (int ***)malloc(CSize * sizeof(int));
    for(int i=0; i<dimension0C;i++) {
        C[i] = (int **)malloc(n * 2 * sizeof(int));
        for(int j=0; j<2; j++) {
            C[i][j] = (int *)malloc(n * sizeof(int));
        }
    }

    for(int i=1; i<n; i++) {
        C[1<<i][0][i] = dist[0*n+i];
        C[1<<i][1][i] = 0;
    }

    fillCData data;
    data.C = C;
    data.CSize = dimension0C * n * 2;
    data.n = n;
    for(int subsetSize=2; subsetSize<n; subsetSize++){
        data.subsetSize = subsetSize;
        #pragma omp parallel
        #pragma omp single
        generateCombinationsAndFillC(NULL, n-1, subsetSize, data);
    }

    // We're interested in all bits but the least significant (the start state)
    int bits = (myPow(2, n) - 1) - 1;

    // Calculate optimal cost
    int opt = -1;
    int parent;
    for(int k=1; k<n; k++)  {
        int cost = C[bits][0][k] + dist[k*n + 0];
        if(opt== -1 || cost<opt){
            opt = cost;
            parent = k;
        }
    }

    for(int i=n-1; i>0; i--) {
        path[i] = parent;
        int new_bits = bits & ~(1 << parent);
        parent = C[bits][1][parent];
        bits = new_bits;
    }

    path[0] = 0;
    free(C);
    return opt;
}

int readMatrix(int size, int **a, const char* filename)
{
    FILE *pf;
    pf = fopen (filename, "r");
    if (pf == NULL)
        return 0;

    for(int i = 0; i < size; ++i)
    {
        for(int j = 0; j < size; ++j) {
            
            fscanf(pf, "%d", a[i] + j);
        }
    }

    fclose (pf);
    return 1;
}

int generateMatrix(int size, int **a)
{
    srand(time(NULL)); 
    int min = 1;
    int max = 1000;
    for(int i = 0; i < size; ++i)
    {
        for(int j = 0; j < size; ++j) {
            if(i==j)
                a[i][j] = 0;
            else
                a[i][j] = rand() % (max - min + 1) + min;
        }
    }
    return 1;
}


int main(int argc, char *argv[])
{
    if(argc<3){
        printf("Type ./filename matrix_size file_with_matrix");
        return -1;
    }

    const int N = atoi(argv[1]);
    int final_path[N];

    // Adjacency matrix for the given graph
    int **adj;
    adj = (int **)malloc(N * N * sizeof(int));
    for(int i=0;i<N;i++){
        adj[i] = (int *)malloc(N * sizeof(int));
    }
    readMatrix(N, adj, argv[2]);
    // generateMatrix(N,adj);

    int* flatAdj = new int[N*N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            flatAdj[i * N + j] = adj[i][j];
        }
    }

    clock_t start, end;
    double cpu_time_used;

    start = clock();
    int final_res = TSP(N, flatAdj, final_path);
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time : %f\n", cpu_time_used);
    printf("Minimum cost : %d\n", final_res);
    printf("Path Taken : ");
    for (int i=0; i<N; i++)
        printf("%d ", final_path[i]);
    printf("\n");
    free(adj);
    return 0;
}